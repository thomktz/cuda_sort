#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "utils.cpp"

void testCUDA(hipError_t error, const char* file, int line) {
  // To catch errors

	if (error != hipSuccess) {
		printf("There is an error in file %s at line %d\n", file, line);
		exit(EXIT_FAILURE);
	}
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

__global__ void merge_k(int *A, int sizeA, int *B, int sizeB, int *M) {
  // Find value for M[i], for thread i.

  int i = threadIdx.x;
  int Kx, Ky, Py, offset, Qx, Qy;

  // No need to define Px, since it's never used
  if (i>sizeA) {
    Kx = i-sizeA;
    Ky = sizeA;
    Py = i-sizeA;
  } else {
    Kx = 0;
    Ky = i;
    Py = 0;
  }

  while (true){
    offset = abs(Ky-Py)/2;
    Qx = Kx+offset;
    Qy = Ky-offset;

    if (
      (Qy >= 0) && (Qx <= sizeB) && ((Qy == sizeA) || (Qx == 0) || (A[Qy] > B[Qx - 1]))
    ) {
      if (
        (Qx == sizeB) || (Qy == 0) || (A[Qy - 1] <= B[Qx])
      ) {
        if (
          (Qy < sizeA) && ((Qx == sizeB) || (A[Qy] <= B[Qx]))
        ) {
          M[i] = A[Qy];
        } else {
          M[i] = B[Qx];
        }
        break;
      } else {
        Kx = Qx+1;
        Ky = Qy-1;
      }
    } else {
      Py = Qy+1;
    }
  }
}

int main(void) {
  // Main script to merge two sorted arrays

  int *A, *B, *M;
  int *A_gpu, *B_gpu, *M_gpu;
  int max_value = 200;
  int sizeA = 10;
  int sizeB = 12; // Different sizes
  int sizeMax = max(sizeA, sizeB);
  int sizeM = sizeA + sizeB;

  A = (int*)malloc(sizeA * sizeof(int));
  B = (int*)malloc(sizeB * sizeof(int));
	M = (int*)malloc(sizeM * sizeof(int));

  hipMalloc(&A_gpu, sizeA * sizeof(int));
  hipMalloc(&B_gpu, sizeB * sizeof(int));
  hipMalloc(&M_gpu, sizeM * sizeof(int));

  generateRandomSortedArray(A, max_value, sizeA);
  generateRandomSortedArray(B, max_value, sizeB);

  hipMemcpy(A_gpu, A, sizeA * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(B_gpu, B, sizeB * sizeof(int), hipMemcpyHostToDevice);

	merge_k <<<1, sizeM>>> (A_gpu, sizeA, B_gpu, sizeB, M_gpu);


  hipMemcpy(M, M_gpu, sizeM * sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "Array A: ";
  printArray(A, sizeA);
  std::cout << "Array B: ";
  printArray(B, sizeB);

  std::cout << "Merged array: ";
  printArray(M, sizeM);

  hipFree(A_gpu);
  hipFree(B_gpu);
  hipFree(M_gpu);
  free(A);
  free(B);
  free(M);

	return 0;
}
