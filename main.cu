#include "hip/hip_runtime.h"
#include <algorithm>
#include "large_merge_path.cu"
#include "utils.cpp"

int main(void) {
    // Main script to merge two sorted arrays

    int *A, *B, *M;
    int *A_gpu, *B_gpu, *M_gpu;
    int max_value = 2000;
    int sizeA = 100;
    int sizeB = 120; // Different sizes
    int sizeMax = max(sizeA, sizeB);
    int sizeM = sizeA + sizeB;
    int blockSize = 1024;
    
    // Pick adequate number of blocks and threads
    int nThreads = std::min(sizeM, blockSize);
    int nBlocks =  (sizeM + blockSize - 1) / blockSize;

    A = (int*)malloc(sizeA * sizeof(int));
    B = (int*)malloc(sizeB * sizeof(int));
    M = (int*)malloc(sizeM * sizeof(int));

    hipMalloc(&A_gpu, sizeA * sizeof(int));
    hipMalloc(&B_gpu, sizeB * sizeof(int));
    hipMalloc(&M_gpu, sizeM * sizeof(int));

    generateRandomSortedArray(A, max_value, sizeA);
    generateRandomSortedArray(B, max_value, sizeB);

    hipMemcpy(A_gpu, A, sizeA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, sizeB * sizeof(int), hipMemcpyHostToDevice);

    parallel_partition <<<nBlocks, nThreads>>> (A_gpu, sizeA, B_gpu, sizeB, M_gpu);

    hipMemcpy(M, M_gpu, sizeM * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Array A: ";
    printArray(A, sizeA);
    std::cout << "Array B: ";
    printArray(B, sizeB);

    std::cout << "Merged array: ";
    printArray(M, sizeM);

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(M_gpu);
    free(A);
    free(B);
    free(M);

    return 0;
}